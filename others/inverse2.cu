#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(err) if (err != hipSuccess) { printf("CUDA Error: %s\n", hipGetErrorString(err)); return -1; }
#define CUBLAS_CHECK(err) if (err != HIPBLAS_STATUS_SUCCESS) { printf("cuBLAS Error\n"); return -1; }

int main() {
    const int n = 3; // Dimensione di ogni matrice
    const int batchSize = 3; // Numero di matrici

    // Host input: 3 matrici (n x n)
    float h_matrices[batchSize][n][n] = {
        { {4, 2, 1}, {2, 5, 3}, {1, 3, 6} }, // Matrice 1
        { {3, 1, 2}, {1, 4, 1}, {2, 1, 3} }, // Matrice 2
        { {2, 1, 1}, {1, 3, 2}, {1, 2, 4} }  // Matrice 3
    };

    // Allocazione in GPU
    float* d_matrices;
    CUDA_CHECK(hipMalloc((void**)&d_matrices, batchSize * n * n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_matrices, h_matrices, batchSize * n * n * sizeof(float), hipMemcpyHostToDevice));

    // Array di puntatori GPU per batched operations
    float** d_matrixArray;
    CUDA_CHECK(hipMalloc((void**)&d_matrixArray, batchSize * sizeof(float*)));
    for (int i = 0; i < batchSize; ++i) {
        float* matrixAddress = d_matrices + i * n * n; // Calcola indirizzo per ogni matrice
        CUDA_CHECK(hipMemcpy(d_matrixArray + i, &matrixAddress, sizeof(float*), hipMemcpyHostToDevice));
    }

    // Array di pivot e info
    int* d_pivotArray;
    int* d_infoArray;
    CUDA_CHECK(hipMalloc((void**)&d_pivotArray, batchSize * n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_infoArray, batchSize * sizeof(int)));

    // Handle di cuBLAS
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Fattorizzazione LU per batched matrices
    CUBLAS_CHECK(hipblasSgetrfBatched(handle, n, d_matrixArray, n, d_pivotArray, d_infoArray, batchSize));

    // Calcolo dell'inversa per batched matrices
    CUBLAS_CHECK(hipblasSgetriBatched(handle, n, (const float**)d_matrixArray, n, d_pivotArray, d_matrixArray, n, d_infoArray, batchSize));

    // Copia risultato su host
    float h_invMatrices[batchSize][n][n];
    CUDA_CHECK(hipMemcpy(h_invMatrices, d_matrices, batchSize * n * n * sizeof(float), hipMemcpyDeviceToHost));

    // Stampa delle matrici inverse
    for (int k = 0; k < batchSize; ++k) {
        printf("Inversa della matrice %d:\n", k + 1);
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) {
                printf("%f ", h_invMatrices[k][i][j]);
            }
            printf("\n");
        }
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_matrices));
    CUDA_CHECK(hipFree(d_matrixArray));
    CUDA_CHECK(hipFree(d_pivotArray));
    CUDA_CHECK(hipFree(d_infoArray));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return 0;
}