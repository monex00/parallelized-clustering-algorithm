#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(err) if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
    exit(EXIT_FAILURE); \
}

#define CUBLAS_CHECK(err) if (err != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS error at line %d\n", __LINE__); \
    exit(EXIT_FAILURE); \
}

template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

template <>
struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

__global__ void reduceWeightMean(
    double* local_means, double* local_weights, double* means, double* weights, int d, int k, int N, int elemsPerThread
) {
    // create shared memory for local means and weights
    double * s_means = SharedMemory<double>();
    double * s_weights = SharedMemory<double>();

    int cluster = blockIdx.x;
    unsigned int startIdx = threadIdx.x * elemsPerThread;

    double sumWeights = 0.0;
    double sumMeans[32] = {0.0};
    // [1 , 2, 3, 4]
    for (unsigned int offset = 0; offset < elemsPerThread; ++offset) {
        int idx = startIdx + offset;
        if (idx < N) {
            sumWeights += local_weights[idx * k + cluster]; // [idx][cluster]
            for (int i = 0; i < d; ++i) {
                sumMeans[i] += local_means[idx * k * d + cluster * d + i];
            }
        }
    }

    s_weights[threadIdx.x] = sumWeights;

    for (int i = 0; i < d; ++i) {
        s_means[threadIdx.x * d + i] = sumMeans[i];
    }

    __syncthreads();


    // reduce local means and weights
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            s_weights[threadIdx.x] += s_weights[threadIdx.x + s];
            for (int i = 0; i < d; ++i) {
                s_means[threadIdx.x * d + i] += s_means[(threadIdx.x + s) * d + i];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        weights[cluster] = s_weights[0] /* / N */;
        for (int i = 0; i < d; ++i) {
            means[cluster * d + i] = s_means[i] / s_weights[0];
        }
    }
}

__global__ void computeResponsibilities(
    const double* data, const double* means, const double* invCovMatrices,
    const double* determinants, const double* weights,
    double* responsibilities, double* local_means, double* local_weights, int d, int k, int N) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx >= N) return;

    // reset local means and weights
    for(int i = 0; i < k; ++i) {
        for(int j = 0; j < d; ++j) {
            local_means[idx * k * d + i * d + j] = 0.0; //[idx][i][j]
        }
        local_weights[idx * k + i] = 0.0; //[idx][i]
    }

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        double sum = 0.0; 
        double diff[32];
        double temp[32];

        for (int cluster = 0; cluster < k; ++cluster) {
            // Calcola la differenza data - mean per il cluster
            for (int j = 0; j < d; ++j) {
                diff[j] = data[i * d + j] - means[cluster * d + j];
            }

            // Calcola il prodotto invCovMatrix * diff
            for (int j = 0; j < d; ++j) {
                temp[j] = 0.0;
                for (int l = 0; l < d; ++l) {
                    temp[j] += invCovMatrices[cluster * d * d + j * d + l] * diff[l];
                }
            }

            // Calcola la distanza di Mahalanobis
            double mahalanobis = 0.0;
            for (int j = 0; j < d; ++j) {
                mahalanobis += diff[j] * temp[j];
            }

            // Calcola la verosimiglianza
            double likelihood = exp(-0.5 * mahalanobis) / 
                                sqrt(pow(2 * M_PI, d) * determinants[cluster]);

            // Calcola la responsabilità pesata
            responsibilities[i * k + cluster] = weights[cluster] * likelihood;
            sum += responsibilities[i * k + cluster];
        }

        // Normalizzazione delle responsabilità
        for (int cluster = 0; cluster < k; ++cluster) {
            // if sum is near 0, set the responsibility to 1/k
            if (sum == 0) {
                responsibilities[i * k + cluster] = 1.0 / k;
            } else {
                responsibilities[i * k + cluster] /= sum;
            }
            local_weights[idx * k + cluster] += responsibilities[i * k + cluster]; //[idx][cluster]

            for (int j = 0; j < d; ++j) {
                local_means[idx * k * d + cluster * d + j] += responsibilities[i * k + cluster] * data[i * d + j]; //[idx][cluster][j]
            }
        }
         
    }
}

__global__ void mStep(
    const double* data, const double* responsibilities, double* means,
    double* local_cov_matrixes, int d, int k, int N) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for  (int cluster = 0; cluster < k; cluster++){
        for(int j = 0; j < d; j++){
            for(int l = 0; l < d; l++){
                local_cov_matrixes[idx * k * d * d + cluster * d * d + j * d + l] = 0.0; //[idx][cluster][j][l]
            }
        }
    }

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        for  (int cluster = 0; cluster < k; cluster++){
            double r = responsibilities[i * k + cluster];
            for(int j = 0; j < d; j++){
                for(int l = 0; l < d; l++){
                    double diff_j = data[i * d + j] - means[cluster * d + j];
                    double diff_l = data[i * d + l] - means[cluster * d + l];
                    local_cov_matrixes[idx * k * d * d + cluster * d * d + j * d + l] += r * diff_j * diff_l; // [idx][cluster][j][l]
                    // printf("Local cov matrixes %d: %f\n", i * k * d * d + cluster * d * d + j * d + l, local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]);
                }
            }
        }
    }    
}

/* __global__ void mStep(
    const double* data, const double* responsibilities, double* means,
    double* covMatrices, double* weights, int d, int k, int N) {

    int cluster = blockIdx.x * blockDim.x + threadIdx.x;
    if (cluster >= k) return;

    double weightSum = weights[cluster];

    for (int i = 0; i < d * d; ++i) {
        covMatrices[cluster * d * d + i] = 0.0; // [cluster][d][i]
    }

    for (int idx = 0; idx < N; ++idx) {
        double r = responsibilities[idx * k + cluster];
        for (int i = 0; i < d; ++i) {
            for (int j = 0; j < d; ++j) {
                double diff_i = data[idx * d + i] - means[cluster * d + i];
                double diff_j = data[idx * d + j] - means[cluster * d + j];
                covMatrices[cluster * d * d + i * d + j] += r * diff_i * diff_j;
            }
        }
    }

   for (int i = 0; i < d; ++i) {
        for (int j = 0; j < d; ++j) {
            covMatrices[cluster * d * d + i * d + j] /= weightSum;
            // Aggiungi il termine di regolarizzazione alla diagonale
            if (i == j) {
                covMatrices[cluster * d * d + i * d + j] += 0.0001;
            }
        }
    }

    weights[cluster] = weights[cluster] / N;
} */

void computeInverseMatrices(
    hipblasHandle_t handle, double* d_matrices, int d, int batchSize,
    double* d_invMatrices, double* d_determinants) {

    double** d_matrixArray;
    CUDA_CHECK(hipMalloc((void**)&d_matrixArray, batchSize * sizeof(double*)));
    double** d_invMatrixArray;
    CUDA_CHECK(hipMalloc((void**)&d_invMatrixArray, batchSize * sizeof(double*)));

    for (int i = 0; i < batchSize; ++i) {
        double* matrixAddress = d_matrices + i * d * d;
        double* invMatrixAddress = d_invMatrices + i * d * d;

        CUDA_CHECK(hipMemcpy(d_matrixArray + i, &matrixAddress, sizeof(double*), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_invMatrixArray + i, &invMatrixAddress, sizeof(double*), hipMemcpyHostToDevice));
    }

    int* d_pivotArray;
    int* d_infoArray;
    CUDA_CHECK(hipMalloc((void**)&d_pivotArray, batchSize * d * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_infoArray, batchSize * sizeof(int)));

    CUBLAS_CHECK(hipblasDgetrfBatched(handle, d, d_matrixArray, d, d_pivotArray, d_infoArray, batchSize));
    
    int * h_pivotArray = (int*)malloc(batchSize * d * sizeof(int));
    double * h_matrixArray = (double*)malloc(batchSize * d * d * sizeof(double));
    double * h_determinants = (double*)malloc(batchSize * sizeof(double));

    hipMemcpy(h_pivotArray, d_pivotArray, batchSize * d * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_matrixArray, d_matrices, batchSize * d * d * sizeof(double), hipMemcpyDeviceToHost);
    

    for (int i = 0; i < batchSize; i++) {
        double det = 1.0f;  // Inizializza a 1.0 per il prodotto
        int swaps = 0;

        for (int j = 0; j < d; j++) {
            // Moltiplicazione di tutti gli elementi diagonali
            det *= h_matrixArray[i * d * d + j * d + j]; // [i][j][j]

            // Controlla se il pivot non è nella posizione attesa
            if (h_pivotArray[i * d + j] != j + 1) { // [i][j]
                swaps++;
            }
        }

        // Cambia segno al determinante se il numero di scambi è dispari
        if (swaps % 2 != 0) {
            det = -det;
        }

        h_determinants[i] = det;
        // printf("Determinante %d: %f\n", i + 1, h_determinants[i]);
    }

    // Copia i determinanti sul dispositivo
    hipMemcpy(d_determinants, h_determinants, batchSize * sizeof(double), hipMemcpyHostToDevice);

    CUBLAS_CHECK(hipblasDgetriBatched(handle, d, (const double**)d_matrixArray, d, d_pivotArray, d_invMatrixArray, d, d_infoArray, batchSize));

   /*  
    printf("Matrici inverse:\n");
    double * h_invMatrixArray = (double*)malloc(batchSize * d * d * sizeof(double));
    cudaMemcpy(h_invMatrixArray, d_invMatrices, batchSize * d * d * sizeof(double), cudaMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++) {
        printf("Matrice %d:\n", i + 1);
        for (int j = 0; j < d; j++) {
            for (int k = 0; k < d; k++) {
                printf("%f ", h_invMatrixArray[i * d * d + j * d + k]);
            }
            printf("\n");
        }
    } */


    free(h_pivotArray);
    free(h_matrixArray);
    free(h_determinants);
    hipFree(d_matrixArray);
    hipFree(d_invMatrixArray);
    hipFree(d_pivotArray);
    hipFree(d_infoArray);
}

void checkCudaError(const char* message) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in %s: %s\n", message, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

double squaredEuclideanDistance(const double *point1, const double *point2, int d)
{
    double distance = 0.0;
    for (int i = 0; i < d; ++i)
    {
        double diff = point1[i] - point2[i];
        distance += diff * diff;
    }
    return distance;
}

// Funzione per inizializzare le medie con k-means++
void initializeMeans(double *h_data, double *h_means, int N, int d, int k)
{
    int *chosenIndices = (int *)malloc(k * sizeof(int));
    double *distances = (double *)malloc(N * sizeof(double));

    // Scegli il primo centro casualmente
    chosenIndices[0] = rand() % N;
    for (int i = 0; i < d; ++i)
    {
        h_means[i] = h_data[chosenIndices[0] * d + i];
    }

    // Inizializza le distanze
    for (int i = 0; i < N; ++i)
    {
        distances[i] = squaredEuclideanDistance(h_data + i * d, h_means, d);
    }

    // Scegli gli altri centri
    for (int cluster = 1; cluster < k; ++cluster)
    {
        double totalDistance = 0.0;
        for (int i = 0; i < N; ++i)
        {
            totalDistance += distances[i];
        }

        // Seleziona il prossimo centro basato sulla probabilità
        double r = ((double)rand() / RAND_MAX) * totalDistance;
        double cumulativeDistance = 0.0;
        int chosenIndex = -1;
        for (int i = 0; i < N; ++i)
        {
            cumulativeDistance += distances[i];
            if (cumulativeDistance >= r)
            {
                chosenIndex = i;
                break;
            }
        }
        chosenIndices[cluster] = chosenIndex;
        for (int i = 0; i < d; ++i)
        {
            h_means[cluster * d + i] = h_data[chosenIndices[cluster] * d + i];
        }

        // Aggiorna le distanze
        for (int i = 0; i < N; ++i)
        {
            double distance = squaredEuclideanDistance(h_data + i * d, h_means + cluster * d, d);
            if (distance < distances[i])
            {
                distances[i] = distance;
            }
        }
    }

    free(chosenIndices);
    free(distances);
}

int main() {
    const int d = 10;    // Numero di features
    const int k = 5;    // Numero di cluster
    const int maxIter = 5;
    const char* fileName = "../data/1M.csv"; // Nome del file CSV
    int threadsPerBlock = 256;
    int dataPerThread = 100;
    const double epsilon = 1e-9;
    double maxChange = 0.0;


    FILE* file = fopen(fileName, "r");
    if (file == NULL) {
        perror("Errore nell'apertura del file CSV");
        return EXIT_FAILURE;
    }

    int N = 0;
    char line[1024];
    while (fgets(line, sizeof(line), file)) {
        N++;
    }

    double* h_data = (double*)malloc(N * d * sizeof(double));
    if (h_data == NULL) {
        perror("Errore nell'allocazione della memoria per i dati");
        fclose(file);
        return EXIT_FAILURE;
    }

    rewind(file);
    int i = 0;
    while (fgets(line, sizeof(line), file)) {
        char* token = strtok(line, ",");
        for (int j = 0; j < d; ++j) {
            if (token != NULL) {
                h_data[i * d + j] = atof(token);
                token = strtok(NULL, ",");
            } else {
                fprintf(stderr, "Errore nella lettura del file CSV alla riga %d\n", i + 1);
                free(h_data);
                fclose(file);
                return EXIT_FAILURE;
            }
        }
        i++;
    }
    fclose(file);

    int totalThreads = (N + dataPerThread - 1) / dataPerThread;
    int numBlocks = (N + threadsPerBlock * dataPerThread - 1) / (threadsPerBlock * dataPerThread);

    double* h_means = (double*)malloc(k * d * sizeof(double));
    // double* h_local_means = (double*)malloc(numBlocks* threadsPerBlock * k * d * sizeof(double));
    double* h_covMatrices = (double*)malloc(k * d * d * sizeof(double));
    double* h_weights = (double*)malloc(k * sizeof(double));
    double* prev_h_means = (double*)malloc(k * d * sizeof(double));
    double* prev_h_covMatrices = (double*)malloc(k * d * d * sizeof(double));
    double* prev_h_weights = (double*)malloc(k * sizeof(double));
    // double* h_local_weights = (double*)malloc(threadsPerBlock * numBlocks * k * sizeof(double));
    // double* h_local_cov_matrixes = (double*)malloc(threadsPrBlock * numBlocks * k * d * d * sizeof(double));

  /*   double feature_means[d];
    for (int j = 0; j < d; ++j) {
        feature_means[j] = 0.0;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < d; ++j) {
            feature_means[j] += h_data[i * d + j];
        }
    }

    for (int j = 0; j < d; ++j) {
        feature_means[j] /= N;
    }

    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < d; ++j) {
            h_means[i * d + j] = feature_means[j] + (double)(rand() % 100) / 100.0; // 5 + (0, 1)
        }
    } */
 initializeMeans(h_data, h_means, N, d, k);

    for (int i = 0; i < k; ++i) {
        h_weights[i] = 1.0 / k;
        for (int j = 0; j < d; ++j) {
            for (int l = 0; l < d; ++l) {
                h_covMatrices[i * d * d + j * d + l] = (j == l) ? 1.0 : 0.0;
            }
        }
    }

    // print intial means
    printf("Means:\n");
    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < d; ++j) {
            printf("%.9f ", h_means[i * d + j]);
        }
        printf("\n");
    }

    double *d_data, *d_means, *d_covMatrices, *d_weights, *d_responsibilities, *d_invCovMatrices, *d_determinants, *d_local_means, *d_local_weights, *d_local_cov_matrixes;
    CUDA_CHECK(hipMalloc(&d_data, N * d * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_means, k * d * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_covMatrices, k * d * d * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_weights, k * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_responsibilities, N * k * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_invCovMatrices, k * d * d * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_determinants, k * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_local_means, k * d * threadsPerBlock * numBlocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_local_weights, k * threadsPerBlock * numBlocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_local_cov_matrixes, k * d * d * threadsPerBlock * numBlocks * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_data, h_data, N * d * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_means, h_means, k * d * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_covMatrices, h_covMatrices, k * d * d * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights, k * sizeof(double), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    // Avvio temporizzazione totale
    CUDA_CHECK(hipEventRecord(start));

    printf("Numero di blocchi: %d\n", numBlocks);
    printf("Numero di thread per blocco: %d\n", threadsPerBlock);
    printf("Numero totale di thread: %d\n", totalThreads);
    printf("Numero di dati per thread: %d\n", dataPerThread);

        
    for (int iter = 0; iter < maxIter; ++iter) {
        // printf("Iterazione %d\n", iter + 1); 

        computeInverseMatrices(handle, d_covMatrices, d, k, d_invCovMatrices, d_determinants);
        hipDeviceSynchronize();
 
        computeResponsibilities<<<numBlocks, threadsPerBlock/* (N + 255) / 256, 256 */>>>(
            d_data, d_means, d_invCovMatrices, d_determinants, d_weights,
            d_responsibilities, d_local_means, d_local_weights, d, k, N);
        hipDeviceSynchronize();

        // copia means e weights e covmatrix h nei valori precedenti
        memcpy(prev_h_means, h_means, k * d * sizeof(double));
        memcpy(prev_h_covMatrices, h_covMatrices, k * d * d * sizeof(double));
        memcpy(prev_h_weights, h_weights, k * sizeof(double));      
         
        

        // copy back local means and weights
        double* h_local_means = (double*)malloc(numBlocks * threadsPerBlock * k * d * sizeof(double));
        double* h_local_weights = (double*)malloc(numBlocks * threadsPerBlock * k * sizeof(double));
        CUDA_CHECK(hipMemcpy(h_local_means, d_local_means, numBlocks * threadsPerBlock * k * d * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_local_weights, d_local_weights, numBlocks * threadsPerBlock * k * sizeof(double), hipMemcpyDeviceToHost));

        // sum local means and weights
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                h_means[i * d + j] = 0.0;
            }
            h_weights[i] = 0.0;
        }

        for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int j = 0; j < k; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_means[j * d + l] += h_local_means[i * k * d + j * d + l];
                }
                h_weights[j] += h_local_weights[i * k + j];
            }
        }

        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                h_means[i * d + j] /= h_weights[i];
            }
            // h_weights[i] /= N;
        }

        CUDA_CHECK(hipMemcpy(d_means, h_means, k * d * sizeof(double), hipMemcpyHostToDevice));
        free(h_local_means);

        // // print responsabilities
        // double* h_responsibilities = (double*)malloc(N * k * sizeof(double));
        // CUDA_CHECK(cudaMemcpy(h_responsibilities, d_responsibilities, N * k * sizeof(double), cudaMemcpyDeviceToHost));

        // printf("Responsabilities:\n");
        // for (int i = 0; i < N; ++i) {
        //     for (int j = 0; j < k; ++j) {
        //         printf("%f ", h_responsibilities[i * k + j]);
        //     }
        //     printf("\n");
        // }
        // printf("\n\n\n");

        // free(h_responsibilities);

        mStep<<<numBlocks, threadsPerBlock>>>(
            d_data, d_responsibilities, d_means, d_local_cov_matrixes, d, k, N);
        hipDeviceSynchronize();
        checkCudaError("mStep");
        

        
        // copy back local cov matrixes 
        double* h_local_cov_matrixes = (double*)malloc(numBlocks * threadsPerBlock * k * d * d * sizeof(double));
        CUDA_CHECK(hipMemcpy(h_local_cov_matrixes, d_local_cov_matrixes, numBlocks * threadsPerBlock * k * d * d * sizeof(double), hipMemcpyDeviceToHost));

        // printf("Local cov matrixes:\n");
        /*  for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int cluster = 0; cluster < k; ++cluster) {
                for (int j = 0; j < d; ++j) {
                    for (int l = 0; l < d; ++l) {
                        printf("%f ", h_local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]);
                    }
                    printf("\n");
                }
                printf("\n\n");
            }
        }
        printf("\n\n\n"); */

        // reset global cov matrixes
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_covMatrices[i * d * d + j * d + l] = 0.0;
                }
            }
        }

        // sum local cov matrixes
        for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int cluster = 0; cluster < k; ++cluster) {
                for (int j = 0; j < d; ++j) {
                    for (int l = 0; l < d; ++l) {
                        h_covMatrices[cluster * d * d + j * d + l] += h_local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]; //[cluster][j][l] += [i][cluster][j][l]
                    }
                }
            }
        }

        // normalize global cov matrixes
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_covMatrices[i * d * d + j * d + l] /= h_weights[i];
                    if(j == l){
                        h_covMatrices[i * d * d + j * d + l] += 0.0001;
                    }
                }
            }
            h_weights[i] /= N;
        }

        // check if prev local means and weights and covMatrix the diff is less than epsilon
        maxChange = 0.0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                double diff = fabs(h_means[i * d + j] - prev_h_means[i * d + j]);
                if (diff > maxChange) {
                    maxChange = diff;
                }
            }
        }
        
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    double diff = fabs(h_covMatrices[i * d * d + j * d + l] - prev_h_covMatrices[i * d * d + j * d + l]);
                    if (diff > maxChange) {
                        maxChange = diff;
                    }
                }
            }
        }

        for (int i = 0; i < k; ++i) {
            double diff = fabs(h_weights[i] - prev_h_weights[i]);
            if (diff > maxChange) {
                maxChange = diff;
            }
        }

        

        CUDA_CHECK(hipMemcpy(d_covMatrices, h_covMatrices, k * d * d * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_weights, h_weights, k * sizeof(double), hipMemcpyHostToDevice));


        free(h_local_cov_matrixes);
        free(h_local_weights);

        if (maxChange < epsilon) {
            printf("Convergenza raggiunta dopo %d iterazioni\n", iter + 1);
            break;
        }
    }

    // Fine temporizzazione totale
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float totalTime = 0;
    CUDA_CHECK(hipEventElapsedTime(&totalTime, start, stop));
    printf("Total time: %f s\n\n", totalTime / 1000.0);

    CUDA_CHECK(hipMemcpy(h_means, d_means, k * d * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_covMatrices, d_covMatrices, k * d * d * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_weights, d_weights, k * sizeof(double), hipMemcpyDeviceToHost));

    printf("Risultati finali:\n");
    for (int i = 0; i < k; ++i) {
        printf("Cluster %d:\n", i + 1);
        printf("Mean: ");
        for (int j = 0; j < d; ++j) {
            printf("%f ", h_means[i * d + j]);
        }
        printf("\nCovariance Matrix:\n");
        for (int j = 0; j < d; ++j) {
            for (int l = 0; l < d; ++l) {
                printf("%f ", h_covMatrices[i * d * d + j * d + l]);
            }
            printf("\n");
        }
        printf("Weight: %f\n", h_weights[i]);
    }

    FILE* paramFile = fopen("model_params.csv", "w");
    if (paramFile == NULL) {
        perror("Errore nella creazione del file dei parametri");
        return EXIT_FAILURE;
    }

    fprintf(paramFile, "Cluster,Feature,Mean,Covariance,Weight\n");
    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < d; ++j) {
            fprintf(paramFile, "%d,%d,%f,", i, j, h_means[i * d + j]);
            for (int l = 0; l < d; ++l) {
                fprintf(paramFile, "%f ", h_covMatrices[i * d * d + j * d + l]);
            }
            fprintf(paramFile, "%f\n", h_weights[i]);
        }
    }
    fclose(paramFile);

    FILE* respFile = fopen("responsibilities.csv", "w");
    if (respFile == NULL) {
        perror("Errore nella creazione del file delle responsabilità");
        return EXIT_FAILURE;
    }

    fprintf(respFile, "DataPoint,Cluster,Responsibility\n");
    double* h_responsibilities = (double*)malloc(N * k * sizeof(double));
    CUDA_CHECK(hipMemcpy(h_responsibilities, d_responsibilities, N * k * sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < k; ++j) {
            fprintf(respFile, "%d,%d,%f\n", i, j, h_responsibilities[i * k + j]);
        }
    }
    fclose(respFile);
    free(h_responsibilities);

    hipFree(d_data);
    hipFree(d_means);
    hipFree(d_covMatrices);
    hipFree(d_weights);
    hipFree(d_responsibilities);
    hipFree(d_invCovMatrices);
    hipFree(d_determinants);
    hipFree(d_local_means);
    hipFree(d_local_weights);
    hipFree(d_local_cov_matrixes);
    free(prev_h_means);
    free(prev_h_covMatrices);
    free(prev_h_weights);
    free(h_data);
    free(h_means);
    free(h_covMatrices);
    free(h_weights);
    
    hipblasDestroy(handle);

    return 0;
}
