#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Macro per il controllo degli errori CUDA
#define CUDA_CHECK(err) if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
    exit(EXIT_FAILURE); \
}

// Macro per il controllo degli errori cuBLAS
#define CUBLAS_CHECK(err) if (err != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS error at line %d\n", __LINE__); \
    exit(EXIT_FAILURE); \
}

// Funzione per invertire una matrice su GPU
void invertMatrix(hipblasHandle_t handle, float* d_matrix, float* d_invMatrix, int n) {
    int *d_pivotArray, *d_info;
    CUDA_CHECK(hipMalloc((void**)&d_pivotArray, n * sizeof(int))); // Pivot array
    CUDA_CHECK(hipMalloc((void**)&d_info, sizeof(int)));          // Info array

    // Array di puntatori per gestire il batch
    float* d_matrixArray[1] = {d_matrix};
    float* d_invMatrixArray[1] = {d_invMatrix};

    float** d_matrixArrayDevPtr;
    float** d_invMatrixArrayDevPtr;

    CUDA_CHECK(hipMalloc((void**)&d_matrixArrayDevPtr, sizeof(d_matrixArray)));
    CUDA_CHECK(hipMalloc((void**)&d_invMatrixArrayDevPtr, sizeof(d_invMatrixArray)));

    CUDA_CHECK(hipMemcpy(d_matrixArrayDevPtr, d_matrixArray, sizeof(d_matrixArray), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_invMatrixArrayDevPtr, d_invMatrixArray, sizeof(d_invMatrixArray), hipMemcpyHostToDevice));

    // Decomposizione LU
    CUBLAS_CHECK(hipblasSgetrfBatched(handle, n, d_matrixArrayDevPtr, n, d_pivotArray, d_info, 1));

    // Calcolo dell'inversa della matrice
    CUBLAS_CHECK(hipblasSgetriBatched(handle, n, (const float**)d_matrixArrayDevPtr, n, d_pivotArray, d_invMatrixArrayDevPtr, n, d_info, 1));

    // Pulizia della memoria temporanea
    CUDA_CHECK(hipFree(d_pivotArray));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_matrixArrayDevPtr));
    CUDA_CHECK(hipFree(d_invMatrixArrayDevPtr));
}

int main() {
    const int n = 3; // Dimensione della matrice
    float h_matrix[n * n] = {
        1, 0, 0,
        0, 1, 0,
        0, 0, 1
    };

    float h_invMatrix[n * n];

    // Allocazione memoria sulla GPU
    float *d_matrix, *d_invMatrix;
    CUDA_CHECK(hipMalloc((void**)&d_matrix, n * n * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_invMatrix, n * n * sizeof(float)));

    // Copia della matrice dalla CPU alla GPU
    CUDA_CHECK(hipMemcpy(d_matrix, h_matrix, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Inizializzazione di cuBLAS
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Calcolo dell'inversa
    invertMatrix(handle, d_matrix, d_invMatrix, n);

    // Copia della matrice inversa dalla GPU alla CPU
    CUDA_CHECK(hipMemcpy(h_invMatrix, d_invMatrix, n * n * sizeof(float), hipMemcpyDeviceToHost));

    // Stampa del risultato
    printf("Matrice inversa:\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%f ", h_invMatrix[i * n + j]);
        }
        printf("\n");
    }

    // Pulizia
    CUDA_CHECK(hipFree(d_matrix));
    CUDA_CHECK(hipFree(d_invMatrix));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return 0;
}
