#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <float.h>

#define CUDA_CHECK(err) if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
    exit(EXIT_FAILURE); \
}

#define CUBLAS_CHECK(err) if (err != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS error at line %d\n", __LINE__); \
    exit(EXIT_FAILURE); \
}

__global__ void computeResponsibilities(
    const float* data, const float* means, const float* invCovMatrices,
    const float* determinants, const float* weights,
    float* responsibilities, float* local_means, float* local_weights, int d, int k, int N) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx >= N) return;

    // reset local means and weights
    for(int i = 0; i < k; ++i) {
        for(int j = 0; j < d; ++j) {
            local_means[idx * k * d + i * d + j] = 0.0f; //[idx][i][j]
        }
        local_weights[idx * k + i] = 0.0f; //[idx][i]
    }

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        float sum = 0.0f; 
        float diff[32];
        float temp[32];

        for (int cluster = 0; cluster < k; ++cluster) {
            // Calcola la differenza data - mean per il cluster
            for (int j = 0; j < d; ++j) {
                diff[j] = data[i * d + j] - means[cluster * d + j];
            }

            // Calcola il prodotto invCovMatrix * diff
            for (int j = 0; j < d; ++j) {
                temp[j] = 0.0f;
                for (int l = 0; l < d; ++l) {
                    temp[j] += invCovMatrices[cluster * d * d + j * d + l] * diff[l];
                }
            }

            // Calcola la distanza di Mahalanobis
            float mahalanobis = 0.0f;
            for (int j = 0; j < d; ++j) {
                mahalanobis += diff[j] * temp[j];
            }

            // Calcola la verosimiglianza
            float likelihood = expf(-0.5f * mahalanobis) / 
                                sqrtf(powf(2 * M_PI, d) * determinants[cluster]);

            // Calcola la responsabilità pesata
            responsibilities[i * k + cluster] = weights[cluster] * likelihood;
            sum += responsibilities[i * k + cluster];
        }

        // Normalizzazione delle responsabilità
        for (int cluster = 0; cluster < k; ++cluster) {
            // if sum is near 0, set the responsibility to 0
            if (sum == 0) {
                responsibilities[i * k + cluster] = 0.0f;
            } else {
                responsibilities[i * k + cluster] /= sum;
            }
            local_weights[idx * k + cluster] += responsibilities[i * k + cluster]; //[idx][cluster]

            for (int j = 0; j < d; ++j) {
                local_means[idx * k * d + cluster * d + j] += responsibilities[i * k + cluster] * data[i * d + j]; //[idx][cluster][j]
            }
        }
         
    }
}

__global__ void mStep(
    const float* data, const float* responsibilities, float* means,
    float* local_cov_matrixes, int d, int k, int N) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i <N; i += gridDim.x * blockDim.x){
        for  (int cluster = 0; cluster < k; cluster++){
            for(int j = 0; j < d; j++){
                for(int l = 0; l < d; l++){
                    local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l] = 0.0; //[i][cluster][j][l]
                }
            }
        }
    }

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        for  (int cluster = 0; cluster < k; cluster++){
            float r = responsibilities[i * k + cluster];
            for(int j = 0; j < d; j++){
                for(int l = 0; l < d; l++){
                    float diff_j = data[i * d + j] - means[cluster * d + j];
                    float diff_l = data[i * d + l] - means[cluster * d + l];
                    local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l] += r /* * diff_j * diff_l */; //[i][cluster][j][l]
                    // printf("Local cov matrixes %d: %f\n", i * k * d * d + cluster * d * d + j * d + l, local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]);
                }
            }
        }
    }

    // print local cov matrixes
    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        for (int cluster = 0; cluster < k; ++cluster) {
            printf("KERNEL: Local cov matrixes %d:\n", i);
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    printf("%f ", local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]);
                }
                printf("\n");
            }
            printf("\n\n");
        }
    }
}

/* __global__ void mStep(
    const float* data, const float* responsibilities, float* means,
    float* covMatrices, float* weights, int d, int k, int N) {

    int cluster = blockIdx.x * blockDim.x + threadIdx.x;
    if (cluster >= k) return;

    float weightSum = weights[cluster];

    for (int i = 0; i < d * d; ++i) {
        covMatrices[cluster * d * d + i] = 0.0; // [cluster][d][i]
    }

    for (int idx = 0; idx < N; ++idx) {
        float r = responsibilities[idx * k + cluster];
        for (int i = 0; i < d; ++i) {
            for (int j = 0; j < d; ++j) {
                float diff_i = data[idx * d + i] - means[cluster * d + i];
                float diff_j = data[idx * d + j] - means[cluster * d + j];
                covMatrices[cluster * d * d + i * d + j] += r * diff_i * diff_j;
            }
        }
    }

   for (int i = 0; i < d; ++i) {
        for (int j = 0; j < d; ++j) {
            covMatrices[cluster * d * d + i * d + j] /= weightSum;
            // Aggiungi il termine di regolarizzazione alla diagonale
            if (i == j) {
                covMatrices[cluster * d * d + i * d + j] += 0.0001;
            }
        }
    }

    weights[cluster] = weights[cluster] / N;
} */

void computeInverseMatrices(
    hipblasHandle_t handle, float* d_matrices, int d, int batchSize,
    float* d_invMatrices, float* d_determinants) {

    float** d_matrixArray;
    CUDA_CHECK(hipMalloc((void**)&d_matrixArray, batchSize * sizeof(float*)));
    float** d_invMatrixArray;
    CUDA_CHECK(hipMalloc((void**)&d_invMatrixArray, batchSize * sizeof(float*)));

    for (int i = 0; i < batchSize; ++i) {
        float* matrixAddress = d_matrices + i * d * d;
        float* invMatrixAddress = d_invMatrices + i * d * d;

        CUDA_CHECK(hipMemcpy(d_matrixArray + i, &matrixAddress, sizeof(float*), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_invMatrixArray + i, &invMatrixAddress, sizeof(float*), hipMemcpyHostToDevice));
    }

    int* d_pivotArray;
    int* d_infoArray;
    CUDA_CHECK(hipMalloc((void**)&d_pivotArray, batchSize * d * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_infoArray, batchSize * sizeof(int)));

    CUBLAS_CHECK(hipblasSgetrfBatched(handle, d, d_matrixArray, d, d_pivotArray, d_infoArray, batchSize));
    
    int * h_pivotArray = (int*)malloc(batchSize * d * sizeof(int));
    float * h_matrixArray = (float*)malloc(batchSize * d * d * sizeof(float));
    float * h_determinants = (float*)malloc(batchSize * sizeof(float));

    hipMemcpy(h_pivotArray, d_pivotArray, batchSize * d * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_matrixArray, d_matrices, batchSize * d * d * sizeof(float), hipMemcpyDeviceToHost);
    

    for (int i = 0; i < batchSize; i++) {
        float det = 1.0f;  // Inizializza a 1.0 per il prodotto
        int swaps = 0;

        for (int j = 0; j < d; j++) {
            // Moltiplicazione di tutti gli elementi diagonali
            det *= h_matrixArray[i * d * d + j * d + j]; // [i][j][j]

            // Controlla se il pivot non è nella posizione attesa
            if (h_pivotArray[i * d + j] != j + 1) { // [i][j]
                swaps++;
            }
        }

        // Cambia segno al determinante se il numero di scambi è dispari
        if (swaps % 2 != 0) {
            det = -det;
        }

        h_determinants[i] = det;
        // printf("Determinante %d: %f\n", i + 1, h_determinants[i]);
    }

    // Copia i determinanti sul dispositivo
    hipMemcpy(d_determinants, h_determinants, batchSize * sizeof(float), hipMemcpyHostToDevice);

    CUBLAS_CHECK(hipblasSgetriBatched(handle, d, (const float**)d_matrixArray, d, d_pivotArray, d_invMatrixArray, d, d_infoArray, batchSize));

   /*  
    printf("Matrici inverse:\n");
    float * h_invMatrixArray = (float*)malloc(batchSize * d * d * sizeof(float));
    cudaMemcpy(h_invMatrixArray, d_invMatrices, batchSize * d * d * sizeof(float), cudaMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++) {
        printf("Matrice %d:\n", i + 1);
        for (int j = 0; j < d; j++) {
            for (int k = 0; k < d; k++) {
                printf("%f ", h_invMatrixArray[i * d * d + j * d + k]);
            }
            printf("\n");
        }
    } */


    free(h_pivotArray);
    free(h_matrixArray);
    free(h_determinants);
    hipFree(d_matrixArray);
    hipFree(d_invMatrixArray);
    hipFree(d_pivotArray);
    hipFree(d_infoArray);
}

int main() {
    const int d = 10;    // Numero di features
    const int k = 5;    // Numero di cluster
    const int maxIter = 2;
    const char* fileName = "data.csv"; // Nome del file CSV
    int threadsPerBlock = 2;
    int dataPerThread = 5;


    FILE* file = fopen(fileName, "r");
    if (file == NULL) {
        perror("Errore nell'apertura del file CSV");
        return EXIT_FAILURE;
    }

    int N = 0;
    char line[1024];
    while (fgets(line, sizeof(line), file)) {
        N++;
    }

    float* h_data = (float*)malloc(N * d * sizeof(float));
    if (h_data == NULL) {
        perror("Errore nell'allocazione della memoria per i dati");
        fclose(file);
        return EXIT_FAILURE;
    }

    rewind(file);
    int i = 0;
    while (fgets(line, sizeof(line), file)) {
        char* token = strtok(line, ",");
        for (int j = 0; j < d; ++j) {
            if (token != NULL) {
                h_data[i * d + j] = atof(token);
                token = strtok(NULL, ",");
            } else {
                fprintf(stderr, "Errore nella lettura del file CSV alla riga %d\n", i + 1);
                free(h_data);
                fclose(file);
                return EXIT_FAILURE;
            }
        }
        i++;
    }
    fclose(file);

    int totalThreads = (N + dataPerThread - 1) / dataPerThread;
    int numBlocks = (N + threadsPerBlock * dataPerThread - 1) / (threadsPerBlock * dataPerThread);

    float* h_means = (float*)malloc(k * d * sizeof(float));
    // float* h_local_means = (float*)malloc(numBlocks* threadsPerBlock * k * d * sizeof(float));
    float* h_covMatrices = (float*)malloc(k * d * d * sizeof(float));
    float* h_weights = (float*)malloc(k * sizeof(float));
    // float* h_local_weights = (float*)malloc(threadsPerBlock * numBlocks * k * sizeof(float));
    // float* h_local_cov_matrixes = (float*)malloc(threadsPrBlock * numBlocks * k * d * d * sizeof(float));

    float feature_means[d];
    for (int j = 0; j < d; ++j) {
        feature_means[j] = 0.0;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < d; ++j) {
            feature_means[j] += h_data[i * d + j];
        }
    }

    for (int j = 0; j < d; ++j) {
        feature_means[j] /= N;
    }

    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < d; ++j) {
            h_means[i * d + j] = feature_means[j] + (float)(rand() % 100) / 100.0; // 5 + (0, 1)
        }
    }


    for (int i = 0; i < k; ++i) {
        h_weights[i] = 1.0 / k;
        for (int j = 0; j < d; ++j) {
            for (int l = 0; l < d; ++l) {
                h_covMatrices[i * d * d + j * d + l] = (j == l) ? 1.0 : 0.0;
            }
        }
    }

    float *d_data, *d_means, *d_covMatrices, *d_weights, *d_responsibilities, *d_invCovMatrices, *d_determinants, *d_local_means, *d_local_weights, *d_local_cov_matrixes;
    CUDA_CHECK(hipMalloc(&d_data, N * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_means, k * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_covMatrices, k * d * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_weights, k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_responsibilities, N * k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_invCovMatrices, k * d * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_determinants, k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_local_means, k * d * threadsPerBlock * numBlocks * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_local_weights, k * threadsPerBlock * numBlocks * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_local_cov_matrixes, k * d * d * threadsPerBlock * numBlocks * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_data, h_data, N * d * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_means, h_means, k * d * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_covMatrices, h_covMatrices, k * d * d * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights, k * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    // Avvio temporizzazione totale
    CUDA_CHECK(hipEventRecord(start));



    printf("Numero di blocchi: %d\n", numBlocks);
    printf("Numero di thread per blocco: %d\n", threadsPerBlock);
    printf("Numero totale di thread: %d\n", totalThreads);
    printf("Numero di dati per thread: %d\n", dataPerThread);


    for (int iter = 0; iter < maxIter; ++iter) {
        // printf("Iterazione %d\n", iter + 1);

        computeInverseMatrices(handle, d_covMatrices, d, k, d_invCovMatrices, d_determinants);
        hipDeviceSynchronize();
 
        computeResponsibilities<<<numBlocks, threadsPerBlock/* (N + 255) / 256, 256 */>>>(
            d_data, d_means, d_invCovMatrices, d_determinants, d_weights,
            d_responsibilities, d_local_means, d_local_weights, d, k, N);
        hipDeviceSynchronize();

        // copy back local means and weights
        float* h_local_means = (float*)malloc(numBlocks * threadsPerBlock * k * d * sizeof(float));
        float* h_local_weights = (float*)malloc(numBlocks * threadsPerBlock * k * sizeof(float));
        CUDA_CHECK(hipMemcpy(h_local_means, d_local_means, numBlocks * threadsPerBlock * k * d * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_local_weights, d_local_weights, numBlocks * threadsPerBlock * k * sizeof(float), hipMemcpyDeviceToHost));

        // sum local means and weights
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                h_means[i * d + j] = 0.0f;
            }
            h_weights[i] = 0.0f;
        }

        for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int j = 0; j < k; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_means[j * d + l] += h_local_means[i * k * d + j * d + l];
                }
                h_weights[j] += h_local_weights[i * k + j];
            }
        }

        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                h_means[i * d + j] /= h_weights[i];
            }
            // h_weights[i] /= N;
        }

        CUDA_CHECK(hipMemcpy(d_means, h_means, k * d * sizeof(float), hipMemcpyHostToDevice));
        free(h_local_means);

        // print responsabilities
        float* h_responsibilities = (float*)malloc(N * k * sizeof(float));
        CUDA_CHECK(hipMemcpy(h_responsibilities, d_responsibilities, N * k * sizeof(float), hipMemcpyDeviceToHost));

        printf("Responsabilities:\n");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < k; ++j) {
                printf("%f ", h_responsibilities[i * k + j]);
            }
            printf("\n");
        }
        printf("\n\n\n");

        free(h_responsibilities);

        mStep<<<numBlocks, threadsPerBlock>>>(
            d_data, d_responsibilities, d_means, d_local_cov_matrixes, d, k, N);
        hipDeviceSynchronize();

        
        // copy back local cov matrixes
        float* h_local_cov_matrixes = (float*)malloc(numBlocks * threadsPerBlock * k * d * d * sizeof(float));
        CUDA_CHECK(hipMemcpy(h_local_cov_matrixes, d_local_cov_matrixes, numBlocks * threadsPerBlock * k * d * d * sizeof(float), hipMemcpyDeviceToHost));

        // printf("Local cov matrixes:\n");
        for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int cluster = 0; cluster < k; ++cluster) {
                for (int j = 0; j < d; ++j) {
                    for (int l = 0; l < d; ++l) {
                        printf("%f ", h_local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]);
                    }
                    printf("\n");
                }
                printf("\n\n");
            }
        }
        printf("\n\n\n");

        // reset global cov matrixes
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_covMatrices[i * d * d + j * d + l] = 0.0f;
                }
            }
        }

        // sum local cov matrixes
        for (int i = 0; i < numBlocks * threadsPerBlock; ++i) {
            for (int cluster = 0; cluster < k; ++cluster) {
                for (int j = 0; j < d; ++j) {
                    for (int l = 0; l < d; ++l) {
                        h_covMatrices[cluster * d * d + j * d + l] += h_local_cov_matrixes[i * k * d * d + cluster * d * d + j * d + l]; //[cluster][j][l] += [i][cluster][j][l]
                    }
                }
            }
        }

        // normalize global cov matrixes
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < d; ++j) {
                for (int l = 0; l < d; ++l) {
                    h_covMatrices[i * d * d + j * d + l] /= h_weights[i];
                    if(j == l){
                        h_covMatrices[i * d * d + j * d + l] += 0.0001;
                    }
                }
            }
            h_weights[i] /= N;
        }

        CUDA_CHECK(hipMemcpy(d_covMatrices, h_covMatrices, k * d * d * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_weights, h_weights, k * sizeof(float), hipMemcpyHostToDevice));


        free(h_local_cov_matrixes);
        free(h_local_weights);
    }

    // Fine temporizzazione totale
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float totalTime = 0;
    CUDA_CHECK(hipEventElapsedTime(&totalTime, start, stop));
    printf("Total time: %f s\n\n", totalTime / 1000.0);

    CUDA_CHECK(hipMemcpy(h_means, d_means, k * d * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_covMatrices, d_covMatrices, k * d * d * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_weights, d_weights, k * sizeof(float), hipMemcpyDeviceToHost));

    printf("Risultati finali:\n");
    for (int i = 0; i < k; ++i) {
        printf("Cluster %d:\n", i + 1);
        printf("Mean: ");
        for (int j = 0; j < d; ++j) {
            printf("%f ", h_means[i * d + j]);
        }
        printf("\nCovariance Matrix:\n");
        for (int j = 0; j < d; ++j) {
            for (int l = 0; l < d; ++l) {
                printf("%f ", h_covMatrices[i * d * d + j * d + l]);
            }
            printf("\n");
        }
        printf("Weight: %f\n", h_weights[i]);
    }

    hipFree(d_data);
    hipFree(d_means);
    hipFree(d_covMatrices);
    hipFree(d_weights);
    hipFree(d_responsibilities);
    hipFree(d_invCovMatrices);
    hipFree(d_determinants);
    hipFree(d_local_means);
    hipFree(d_local_weights);
    hipFree(d_local_cov_matrixes);
    free(h_data);
    free(h_means);
    free(h_covMatrices);
    free(h_weights);
    
    hipblasDestroy(handle);

    return 0;
}
